#include "hip/hip_runtime.h"
#include <optix_world.h>
using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

struct Vertex{
	float3 position;
	float3 normal;
};

rtBuffer<Vertex> vertex_buffer;
rtBuffer<int3>   index_buffer;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void mesh_intersect(int primIdx){
	int3 v_idx = index_buffer[primIdx];

	float3 p0 = vertex_buffer[v_idx.x].position;
	float3 p1 = vertex_buffer[v_idx.y].position;
	float3 p2 = vertex_buffer[v_idx.z].position;

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {

			float3 n0 = vertex_buffer[v_idx.x].normal;
			float3 n1 = vertex_buffer[v_idx.y].normal;
			float3 n2 = vertex_buffer[v_idx.z].normal;
			shading_normal = normalize(n0*(1.0f - beta - gamma) + n1*beta + n2*gamma);
			geometric_normal = normalize(n);

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void mesh_bounds(int primIdx, optix::Aabb* aabb){
	const int3 v_idx = index_buffer[primIdx];

	const float3 v0 = vertex_buffer[v_idx.x].position;
	const float3 v1 = vertex_buffer[v_idx.y].position;
	const float3 v2 = vertex_buffer[v_idx.z].position;
	const float  area = length(cross(v1 - v0, v2 - v0));

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else {
		aabb->invalidate();
	}
}