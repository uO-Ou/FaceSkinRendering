#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
using namespace optix;

struct PerRayData_shadow{
	float distance;
};

rtDeclareVariable(int, WinWidth, , );
rtDeclareVariable(int, WinHeight, , );

/**************for ray-casting*************/
rtDeclareVariable(float, nearp, , );
rtDeclareVariable(float2, canvaSize, , );
rtDeclareVariable(float3, camera_x, , );
rtDeclareVariable(float3, camera_y, , );
rtDeclareVariable(float3, camera_z, , );
/******************************************/

//lights buffer
rtBuffer<float, 1> lights_buffer;
rtBuffer<unsigned int, 1> sm_buffer;
rtBuffer<uchar4, 2> output_buffer_uchar4;
rtBuffer<float4, 2> output_buffer_float4;

rtDeclareVariable(float, diff_mix, , );
rtDeclareVariable(float, face_roughness,,);

//gbuffer textures
rtTextureSampler<float4, 2>  position_texture;
rtTextureSampler<float4, 2>  normal_texture;
rtTextureSampler<float4, 2>  diffuse_texture;
rtTextureSampler<float4, 2>  specular_texture;
rtTextureSampler<float4, 2>  reflectance_texture;

rtDeclareVariable(uint, light_cnt, , );
rtDeclareVariable(uint, scr_width, , );
rtDeclareVariable(float3, camera_pos, , );
rtDeclareVariable(float3, bgcolor, , );
rtDeclareVariable(int, option, , );

//others
rtDeclareVariable(uint, shadow_ray_type, , );                    //����ray��type
rtDeclareVariable(float, scene_epsilon, , );                     //epsilon
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );         //launch_index  

rtDeclareVariable(rtObject, shadow_casters, , );

rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );   //����ray��Я������
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );       //�����������Ϊt_hit


RT_PROGRAM void any_hit_shadow(){
	prd_shadow.distance = t_hit;
	rtTerminateRay();
}

RT_PROGRAM void rayCasting(){
	float tx = ((0.5f + launch_index.x) / WinWidth - 0.5f) * canvaSize.x;
	float ty = ((0.5f + launch_index.y) / WinHeight - 0.5f) * canvaSize.y;
	float3 direction = camera_x*tx + camera_y*ty + camera_z*nearp;
	
	PerRayData_shadow prd;  prd.distance = -1;
	optix::Ray ray = optix::make_Ray(camera_pos+direction, normalize(direction), shadow_ray_type, scene_epsilon, 10000);
	rtTrace(shadow_casters, ray, prd);
	if (prd.distance > 0)
		output_buffer_uchar4[launch_index] = make_uchar4(0, 255, 0, 255);
}

/*
option: 0, uchar4
		1, float4
*/

__device__ float fresnelReflectance(float3 halfDir, float3 viewDir, float F0){
	float base = 1.0 - dot(halfDir, viewDir);
	float exponential = powf(base, 5.0);
	return exponential + F0*(1.0 - exponential);
}

__device__ float PHBeckmann(float nDotH, float m){
	float alpha = acos(nDotH);
	float tanAlpha = tan(alpha);
	float value = exp(-(tanAlpha*tanAlpha) / (m*m)) / (m*m*powf(nDotH, 4.0));
	return value;
}

__device__ float brdf_KS(float3 normal, float3 lightDir, float3 viewDir, float roughness, float specPower){
	float result = 0.0;
	float NdotL = dot(normal,lightDir);
	if (NdotL > 0){
		float3 h = lightDir + viewDir;
		float3 halfDir = normalize(h);
		float NdotH = dot(normal,halfDir);
		float PH = PHBeckmann(NdotH,roughness);
		const float F0 = 0.028;
		float F = fresnelReflectance(halfDir,viewDir,F0);
		float frSpec = max(PH*F / dot(h, h), 0.0);
		result = frSpec * NdotL * specPower;
	}
	return result;
}

RT_PROGRAM void shading(){
	float3 norm = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
	float3 shadeResult = make_float3(0.0f, 0.0f, 0.0f);

	float final_spec = 0.0;
	if (dot(norm, norm)>0.0f) {	//valid pixel
		float3 pos = make_float3(tex2D(position_texture, launch_index.x, launch_index.y));
		float3 mat_diff = make_float3(tex2D(diffuse_texture, launch_index.x, launch_index.y));
		
		for (unsigned int lid = 0u; lid < light_cnt; ++lid){
			float3 light_pos = *((float3 *)(&lights_buffer[lid * 9 + 0]));
			float3 light_col = *((float3 *)(&lights_buffer[lid * 9 + 3]));
			//float3 light_atn = *((float3 *)(&lights_buffer[lid * 9 + 6]));

			float3 L = light_pos - pos;
			if (dot(L, norm)>0.0f){
				//distance to light
				float dist = sqrtf(dot(L, L));
				L /= dist;

				//shadow ray. check if light was blocked
				PerRayData_shadow prd;  prd.distance = -1;
				optix::Ray ray = optix::make_Ray(pos, L, shadow_ray_type, scene_epsilon, dist);
				rtTrace(shadow_casters, ray, prd);

				if (prd.distance > 0){
					
				}
				else{
					float NdotL = max(0.0f, dot(L, norm));
					float reflect_ratio = tex2D(reflectance_texture, WinWidth*NdotL, WinHeight*face_roughness).x;
					
					//diffuse
					shadeResult += light_col * mat_diff * NdotL * (1 - reflect_ratio);

					//specular
					float3 viewDir = normalize(camera_pos - pos);
					float spec = brdf_KS(norm, L, viewDir, face_roughness, 1) * reflect_ratio;

					//shadeResult += make_float3(spec*light_col.x, spec*light_col.y, spec*light_col.z);
					final_spec += spec*light_col.x;
				}
			}
		}
		shadeResult += mat_diff * 0.05f;
	}
	else{
		//shadeResult = make_float3(0.498, 0.235, 0.137);
		shadeResult = bgcolor;
	}
	shadeResult.x = clamp(shadeResult.x, 0.0f, 1.0f);
	shadeResult.y = clamp(shadeResult.y, 0.0f, 1.0f);
	shadeResult.z = clamp(shadeResult.z, 0.0f, 1.0f);
	if (option == 0)
		output_buffer_uchar4[launch_index] = make_uchar4(shadeResult.x * 255, shadeResult.y * 255, shadeResult.z * 255, 255);
	else
		output_buffer_float4[launch_index] = make_float4(shadeResult.x, shadeResult.y, shadeResult.z, final_spec);
}

RT_PROGRAM void exception(){

}
